#include <iostream>
#include <fstream>
#include "state.cu"
#include "exist_state.cu"

int main(){
    int n = 30;
    hipSetDevice(1);
    QuantumState state(n);
    exist_QuantumState exist_state(n);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time, exist_time;


    std::ofstream H_ofs("H_gate.csv");
    for(int i = 0; i < n; i++){
        // 提案手法の時間計測
        hipEventRecord(start);
        state.act_H(i);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        // 既存手法の時間計測
        hipEventRecord(start);
        exist_state.act_H(i);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&exist_time, start, stop);

        H_ofs << "H" << i << ", " << time << ", " << exist_time << ", "  << time / exist_time << ", "<< std::endl;
    }

    std::ofstream S_ofs("S_gate.csv");
    for(int i = 0; i < n; i++){
        // 提案手法の時間計測
        hipEventRecord(start);
        state.act_S(i);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        // 既存手法の時間計測
        hipEventRecord(start);
        exist_state.act_S(i);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&exist_time, start, stop);

        S_ofs << "H" << i << ", " << time << ", " << exist_time << ", "  << time / exist_time << ", "<< std::endl;
    }

    std::ofstream T_ofs("T_gate.csv");
    for(int i = 0; i < n; i++){
        // 提案手法の時間計測
        hipEventRecord(start);
        state.act_T(i);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        // 既存手法の時間計測
        hipEventRecord(start);
        exist_state.act_T(i);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&exist_time, start, stop);

        T_ofs << "H" << i << ", " << time << ", " << exist_time << ", "  << time / exist_time << ", "<< std::endl;
    }

    hipEventRecord(start);
    state.act_CNOT(0, 1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    std::cout << "CNOT " << time << std::endl;

    CTYPE* vec = state.get_vec();
    std::cout << vec[0] << std::endl;
    return 0;
}
